#include "sha256.h"
#include <map>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cuda_util.h"

std::map<int, uint32_t *> context_idata[2];
std::map<int, uint32_t *> context_odata[2];
std::map<int, hipStream_t> context_streams[2];
std::map<int, uint32_t *> context_tstate[2];
std::map<int, uint32_t *> context_ostate[2];
std::map<int, uint32_t *> context_hash[2];

#define HASH_SIZE (8 * sizeof(uint32_t))

int main(int argc, const char* argv[]) {
    int thrd_id = 0;
    int throughput = 128;
    int stream = 0;
    uint32_t cpu_midstate[8] = {0};

    if (argc != 4) {
        printf("Usage: gpuverify <num_blocks> <num verify loops> <input>\n");
        return 1;
    }

    fprintf(stderr, "starting:\n");
    int num_blocks = strtol(argv[1], nullptr, 10);
    int num_verify = strtol(argv[2], nullptr, 10);
    int input = strtol(argv[3], nullptr, 10);

    //uint32_t* h_pdata = (uint32_t*)calloc(num_blocks * 16, sizeof(uint32_t));

    context_idata[stream][0] = NULL;
    hipMalloc(&context_idata[stream][0], 32 * sizeof(uint32_t));
    //hipMemset(&context_idata[stream][0], strtol(argv[1], nullptr, 10), 32 * sizeof(uint32_t));

    context_odata[stream][0] = NULL;
    hipMalloc(&context_odata[stream][0], 32 * sizeof(uint32_t));
    //hipMemset(&context_odata[stream][0], strtol(argv[1], nullptr, 10), 32 * sizeof(uint32_t));

    context_ostate[stream][0] = NULL;
    hipMalloc(&context_ostate[stream][0], 32 * sizeof(uint32_t));

    context_tstate[stream][0] = NULL;
    hipMalloc(&context_tstate[0][0], 32 * sizeof(uint32_t));
 
    context_hash[stream][0] = NULL;
    hipMalloc(&context_hash[stream][0], 8 * sizeof(uint32_t));

    uint32_t* d_hash = NULL;
    hipMalloc(&d_hash, num_blocks * 8 * sizeof(uint32_t));

    hipStream_t cudaStream;
    hipStreamCreate(&cudaStream);
    context_streams[stream][0] = cudaStream;

    uint8_t h_pdata[65] = "AnatolyYakovenko11/2/201712pmPSTAnatolyYakovenko11/2/201712pmPST";

    size_t input_size_bytes = num_blocks * 16 * sizeof(uint32_t);
    //memset(h_pdata, input, input_size_bytes);
    uint32_t* d_pdata = nullptr;
    hipMalloc(&d_pdata, input_size_bytes);
    checkCudaErrors(hipMemcpy(d_pdata, h_pdata, input_size_bytes, hipMemcpyHostToDevice));
    //hipMemset(d_pdata, strtol(argv[1], nullptr, 10), 20 * sizeof(uint32_t));
    memset(h_pdata, 0, input_size_bytes);
    hipMemcpy(h_pdata, d_pdata, input_size_bytes, hipMemcpyDeviceToHost);
    for (int i = 0; i < 64/4; i++) {
        printf("%x ", ((uint32_t*)h_pdata)[i]);
    }
    printf("\n");

    printf("starting verify\n");

    uint32_t* h_hash = (uint32_t*)calloc(num_blocks * HASH_SIZE, 1);

    prepare_sha256(thrd_id, cpu_midstate);
    sha256_verify(d_pdata, d_hash, num_blocks, num_verify);

    hipDeviceSynchronize();

    hipMemcpy(h_hash, d_hash, num_blocks * HASH_SIZE, hipMemcpyDeviceToHost);

    for (int i = 0; i < (num_blocks * HASH_SIZE) / sizeof(uint32_t); i++) {
        if ((i != 0) && i % 8 == 0) {
            printf("\n");
        }
        printf("%08x ", h_hash[i]);
    }
    printf("\n");
 
    return 0;
}
